#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include ""

#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <thrust\device_vector.h>
#include <thrust\device_ptr.h>
#include <time.h>
#include <limits>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#include "global.h"
#include "ray.h"
#include "camera.h"

#include "object\objectList.h"
#include "object\sphere.h"
#include "object\sphereMoving.h"
#include "object\BVHnode.h"

#include "material\lambertian.h"
#include "material\matel.h"
#include "material\dielectric.h"

#include "texture\texture.h"
#include "texture\CheckerTexture.h"
#include "texture\ImageTexture.h"

#define CHECK_CUDA_ERRORS(val) checkCuda( (val), #val, __FILE__, __LINE__)

void checkCuda(hipError_t result, char const *const func, const char* const file, int const line){
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << " :" << line << "'" << func << "'\n";
        hipDeviceReset();
        system("pause");
        exit(99);
    }
}

__device__ Vec3 rayColor(const Ray& r, Object** world, int maxDepth, hiprandState* localRandState) {
    Ray currRay = r;
    Vec3 currAttenuation(1.0, 1.0, 1.0);
    for (int i = 0; i < maxDepth; ++i) {
        HitRecord rec;
        if ((*world)->hit(currRay, 0.001, Infinity, rec)) {
            Vec3 attenuation;
            Ray scattered;
            if (rec.mat->scatter(currRay, rec, attenuation, scattered, localRandState)) {
                currAttenuation = currAttenuation * attenuation;
                currRay = scattered;
            }
            else return Vec3(0, 0, 0);
        }
        else {
            Vec3 dir = normalize(r.getDir());
            float t = 0.5 * (dir.getY() + 1);
            return ((1.0 - t) * Vec3(1.0, 1.0, 1.0) + t * Vec3(0.5, 0.7, 1.0)) * currAttenuation;
        }
    }
    return Vec3(0, 0, 0);
}

__global__ void cudaRandInit(hiprandState* randState) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1985, 0, 0, randState);
    }
}

__global__ void renderInit(int width, int height, hiprandState* randState) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= width || j >= height) return;
    int index = j * width + i;
    hiprand_init(1985, index, 0, &randState[index]);
}

#define cuda_rand (hiprand_uniform(&localRandState))

__global__ void render(unsigned char* fb, int width, int height, Camera** cam, Object** world, int samplerPerPixel, int maxDepth, hiprandState* randState) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= width || j >= height) return;

    int index = 4 * (j * width + i);
    Vec3 color(0.0, 0.0, 0.0);
    hiprandState localRandState = randState[index/4];
    for (int k = 0; k < samplerPerPixel; ++k) {
        float u = (i + cuda_rand) / width;
        float v = (j + cuda_rand) / height;
        Ray r = (*cam)->getRay(u, v, &localRandState);
        color += rayColor(r, world, maxDepth, &localRandState);
    }
    randState[index / 4] = localRandState;
    color = color / samplerPerPixel;
    int inverseIndex = 4 * ((height - 1 - j) * width + i);
    fb[inverseIndex + 0] = sqrt(color.getX()) * 255.999;
    fb[inverseIndex + 1] = sqrt(color.getY()) * 255.999;
    fb[inverseIndex + 2] = sqrt(color.getZ()) * 255.999;
    fb[inverseIndex + 3] = 255;
}

__global__ void createWorld(Object** list, Object** world, Camera** cam, int width, int height, hiprandState* randState, int* worldObjNum, unsigned char* earth, int w, int h, int n) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState localRandState = *randState;
        auto checker = new CheckerTexture(new RGBtexture(Vec3(0.2, 0.3, 0.1)), new RGBtexture(Vec3(0.9, 0.9, 0.9)));

        int i = 0;
        list[i++] = new Sphere(Vec3(0.0, -1000.0, 0.0), 1000.0, new Lambertian(checker));
        for (int a = -10; a < 10; ++a) {
            for (int b = -10; b < 10; ++b) {
                float chooseMat = cuda_rand;
                Vec3 center(a + cuda_rand * 0.9, 0.2, b + cuda_rand * 0.9);
                if (chooseMat < 0.5) {
                    list[i++] = new SphereMoving(center, center + Vec3(0, cuda_rand * 0.5, 0), 0.0, 1.0,  0.2, new Lambertian(new RGBtexture(Vec3(cuda_rand * cuda_rand, cuda_rand * cuda_rand, cuda_rand * cuda_rand))));
                }
                else if (chooseMat < 0.85) {
                    list[i++] = new Sphere(center, 0.2, new Matel(Vec3(0.5 * (1.0 + cuda_rand), 0.5 * (1.0 + cuda_rand), 0.5 * (1.0 + cuda_rand)), 0.35 * cuda_rand));
                }
                else {
                    list[i++] = new Sphere(center, 0.2, new Dielectric(1.5));
                }
            }
        }
        list[i++] = new Sphere(Vec3(0, 1.0, 0), 1.0, new Dielectric(1.5));
        list[i++] = new Sphere(Vec3(4, 1.0, 0), 1.0, new Matel(Vec3(0.7, 0.6, 0.5), 0.0));
        list[i++] = new Sphere(Vec3(-4.0, 1.0, 0), 1.0, new Lambertian(new ImageTexture(earth, w, h, n)));
        *worldObjNum = i;

        *randState = localRandState;
        //*world = new BVHnode(list, 0, i, 0.0, 1.0, &localRandState);
        *world = new ObjectList(list, i);

        // camera initialize
        Vec3 cameraPos(17, 2, -1.5);
        Vec3 cameraTarget(0, 0, -0.5);
        float fov = 30.0;
        float focusDistance = 12.0;
        float aperture = 0;
        float beginTime = 0.0;
        float endTime = 1.0;

        *cam = new Camera(cameraPos, cameraTarget, Vec3(0, 1.0, 0), fov, float(width) / float(height), aperture, focusDistance, beginTime, endTime);

        //free(list);
    }
}

__global__ void freeWorld(Object** list, Object** world, Camera** cam, int* objectNum) {
    for (int i = 0; i < *objectNum; ++i) {
        delete ((Sphere*)list[i])->mat;
        delete list[i];
    }
    delete world;
    delete cam;
    delete objectNum;
}

int main()
{
	/*int dev = 0;
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, dev);
    std::cout << "ʹ��GPU device " << dev << ": " << devProp.name << std::endl;
    std::cout << "SM��������" << devProp.multiProcessorCount << std::endl;
    std::cout << "ÿ���߳̿�Ĺ����ڴ��С��" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
    std::cout << "ÿ���߳̿������߳�����" << devProp.maxThreadsPerBlock << std::endl;
    std::cout << "ÿ��EM������߳�����" << devProp.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "ÿ��EM������߳�������" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;*/

    const int maxDepth = 50;
    const int samplerPerPixel = 500;
    const int width = 1920;
    const int height = 1080;

    // image
    const int pixelsNum = width * height;

    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << width << " * " << height << " image ";
    std::cerr << " in " << tx << " * " << ty << " blocks.\n";

    // fb
    size_t fbSize = 4 * pixelsNum * sizeof(unsigned char);

    // allocate fb
    unsigned char* fb;
    CHECK_CUDA_ERRORS(hipMallocManaged((void**)&fb, fbSize));

    // allocate random state
    hiprandState* totalRandState;
    CHECK_CUDA_ERRORS(hipMalloc((void**)&totalRandState, pixelsNum * sizeof(hiprandState)));
    hiprandState* randState;
    CHECK_CUDA_ERRORS(hipMalloc((void**)&randState, sizeof(hiprandState)));

    cudaRandInit<<<1, 1 >>> (randState);
    CHECK_CUDA_ERRORS(hipGetLastError());
    CHECK_CUDA_ERRORS(hipDeviceSynchronize());

    // world & camera data
    Object** list;
    CHECK_CUDA_ERRORS(hipMallocManaged((void**)&list, (20 * 20 + 1 + 3) * sizeof(Object*)));
    Object** world;
    CHECK_CUDA_ERRORS(hipMallocManaged((void**)&world, sizeof(Object*)));
    Camera** cam;
    CHECK_CUDA_ERRORS(hipMallocManaged((void**)&cam, sizeof(Camera*)));
    int* worldObjNum;
    CHECK_CUDA_ERRORS(hipMallocManaged((void**)&worldObjNum, sizeof(int)));

    int w, h, n;
    unsigned char* earthImg = stbi_load("earth.jpg", &w, &h, &n, 0);
    unsigned char* cudaEarth;
    CHECK_CUDA_ERRORS(hipMalloc((void**)&cudaEarth, sizeof(unsigned char) * w * h * n));
    hipMemcpy(cudaEarth, earthImg, w * h * n * sizeof(unsigned char), hipMemcpyHostToDevice);

    *worldObjNum = 0;
    createWorld<<<1, 1 >>> (list, world, cam, width, height, randState, worldObjNum, cudaEarth, w, h, n);
    CHECK_CUDA_ERRORS(hipGetLastError());
    CHECK_CUDA_ERRORS(hipDeviceSynchronize());

    clock_t begin, end;
    std::cerr << "\nStart rendering\n";
    begin = clock();
    // render buffer
    dim3 blocks(width / tx + 1, height / ty + 1);
    dim3 threads(tx, ty);
    renderInit<<<blocks, threads>>> (width, height, totalRandState);
    CHECK_CUDA_ERRORS(hipGetLastError());
    CHECK_CUDA_ERRORS(hipDeviceSynchronize());
    render<<<blocks, threads>>>(fb, width, height, cam, world, samplerPerPixel, maxDepth, totalRandState);
    CHECK_CUDA_ERRORS(hipGetLastError());
    CHECK_CUDA_ERRORS(hipDeviceSynchronize());

    end = clock();
    double time = ((double)(end - begin)) / CLOCKS_PER_SEC;
    std::cerr << "End\ntook " << time << " s.\n";

    /*unsigned char* data[4 * pixelsNum];
    hipMemcpy(fb, data, fbSize, hipMemcpyDeviceToHost);*/

    // output image png
    stbi_write_png("main.png", width, height, 4, fb, width * 4);

    //freeWorld<<<1, 1 >>>(list, world, cam, worldObjNum);
    CHECK_CUDA_ERRORS(hipFree(totalRandState));
    CHECK_CUDA_ERRORS(hipFree(randState));
    CHECK_CUDA_ERRORS(hipFree(worldObjNum));
    CHECK_CUDA_ERRORS(hipFree(list));
    CHECK_CUDA_ERRORS(hipFree(world));
    CHECK_CUDA_ERRORS(hipFree(cam));
    CHECK_CUDA_ERRORS(hipFree(fb));
    hipDeviceReset();
    system("pause");
}